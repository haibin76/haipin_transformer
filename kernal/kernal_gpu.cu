#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#if !_WIN32
#include <sys/time.h>
#endif

#include <hip/hip_runtime.h>
#include "kernal.h"

//�ڴ�GPU��֮ǰ����Ҫͬʱ��ʼ��GPU���鿴GPU��һЩ���ܺͲ�������
bool init_gpu(int device_id)
{
    //�鿴�м���GPU��
    printf("%s Starting...\n", device_id);
    int device_count = 0;
    hipError_t error_id = hipGetDeviceCount(&device_count);
    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    if (device_count == 0) {
        printf("There are no available device(s) that support CUDA\n");
    } else {
        printf("Detected %d CUDA Capable device(s)\n", device_count);
        if (device_count <device_id) {
            printf("the input parameters device_id:%d error\n", device_id);
        }
    }

    int dev, driverVersion = 0, runtimeVersion = 0;
    dev = device_id;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Device %d: \"%s\"\n", dev, deviceProp.name);
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf(" CUDA Driver Version / Runtime Version %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
    printf(" CUDA Capability Major/Minor version number: %d.%d\n", deviceProp.major, deviceProp.minor);
    printf(" Total amount of global memory: %.2f MBytes (%llu bytes)\n", (float)deviceProp.totalGlobalMem/(pow(1024.0,3)), (unsigned long long) deviceProp.totalGlobalMem);
    printf(" GPU Clock rate: %.0f MHz (%0.2f GHz)\n",
    deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
    printf(" Memory Clock rate: %.0f Mhz\n",
    deviceProp.memoryClockRate * 1e-3f);
    printf(" Memory Bus Width: %d-bit\n",
    deviceProp.memoryBusWidth);
    if (deviceProp.l2CacheSize) {
        printf(" L2 Cache Size: %d bytes\n",
        deviceProp.l2CacheSize);
    }
    printf(" Max Texture Dimension Size (x,y,z) 1D=(%d), 2D=(%d,%d), 3D=(%d,%d,%d)\n",
            deviceProp.maxTexture1D,
            deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
            deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);

    printf(" Max Layered Texture Size (dim) x layers 1D=(%d) x %d, 2D=(%d,%d) x %d\n",
        deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1],
        deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
        deviceProp.maxTexture2DLayered[2]);

    printf(" Total amount of constant memory: %lu bytes\n", deviceProp.totalConstMem);
    printf(" Total amount of shared memory per block: %lu bytes\n", deviceProp.sharedMemPerBlock);
    printf(" Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
    printf(" Warp size: %d\n", deviceProp.warpSize);
    printf(" Maximum number of threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
    printf(" Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    printf(" Maximum sizes of each dimension of a block: %d x %d x %d\n",
        deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf(" Maximum sizes of each dimension of a grid: %d x %d x %d\n",
        deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf(" Maximum memory pitch: %lu bytes\n", deviceProp.memPitch);
    exit(EXIT_SUCCESS);
}

__global__ void matrix_create_gpu(float* matrix, int height, int width)
{
    int row = blockIdx.x + threadIdx.x;

    for(int i = 0; i < width; i++) {
        matrix[row * width +i] = (float)(((rand() % 0xFF) - 128) * 1.0 / 256);
    }

    return;
}

//��Host������һ��GPU�ڴ棬����ʹ��GPU���������ʼ��, batch_num >= 32, һ���߳���һ�У�һ��block������32��
float* matrix_create(int batch_num, int height, int width)
{
    float *matrix;
    hipMalloc((void **)(&matrix), batch_num * height * width * sizeof(float));
    if (!matrix)
        return NULL;

    dim3 block(32, 1);
    dim3 grid(batch_num * height / 32, 1);

    matrix_create_gpu<<<grid, block>>>(matrix, height, width);
    return matrix;
}

void matrix_delete(void* matrix)
{
    hipFree(matrix);
    return;
}

__global__ void matrix_add_gpu(int batch_num, float* matrix_a, float* matrix_b, float* matrix_c, char* mask, int height, int width)
{
    //32���߳���һ��
    int thread_len = width / blockDim.x;
    int l_pos = threadIdx.y * width + threadIdx.x * thread_len;
    int g_pos = blockIdx.x * height * width + l_pos;

    for(int i = 0 i < thread_len; i++)
        if (mask && mask[l_pos])
            c[g_pos +i] = a[g_pos +i] + b[g_pos +i];
        else
            c[g_pos +i] = 0;

    return;
}

void matrix_add(int batch_num, float* matrix_a, float* matrix_b, float* matrix_c, char* mask, int height, int width)
{
    dim3 block(32, height, 1);
    dim3 grid(batch_num, 1, 1);

    matrix_add_gpu<<<grid, block>>>(matrix_a, matrix_b, matrix_c, mask, int height, int width);

    return;
}

__global__ void matrix_multi_gpu(int batch_num, float *matrix_a, float *matrix_b, float *matrix_c, int height_a,
                                    int width_a, int height_b,
                                    int width_b, int height_c,
                                    int width_c)
{
    #define LOCAL_BLOCK_SIZE 32
    __shared__ float ds_M[LOCAL_BLOCK_SIZE][LOCAL_BLOCK_SIZE];
    __shared__ float ds_N[LOCAL_BLOCK_SIZE][LOCAL_BLOCK_SIZE];
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * LOCAL_BLOCK_SIZE + ty;
    int Col = bx * LOCAL_BLOCK_SIZE + tx;
    float Pvalue = 0;
    for (int m = 0; m < (width_a - 1) / LOCAL_BLOCK_SIZE + 1; ++m) {
        if (Row < height_a && m * LOCAL_BLOCK_SIZE + tx < width_a) {
            ds_M[ty][tx] = matrix_a[Row * width_a + m * LOCAL_BLOCK_SIZE + tx];
        } else {
            ds_M[ty][tx] = 0.0;
        }

        if (Col < width_b && m * LOCAL_BLOCK_SIZE + ty < height_b) {
            ds_N[ty][tx] = matrix_b[(m * LOCAL_BLOCK_SIZE + ty) * width_b + Col];
        } else {
            ds_N[ty][tx] = 0.0;
        }
        __syncthreads();

        for (int k = 0; k < LOCAL_BLOCK_SIZE; ++k) {
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        }
        __syncthreads();
    }

    if (Row < height_c && Col < width_c) {
        matrix_c[Row * width_c + Col] = Pvalue;
    }
    return;
}

void matrix_multi(int batch_num, float* matrix_a, int a_height, int a_width, float* matrix_b, int b_width, float* matrix_c)
{
    dim3 block(32, 32, 1);
    dim3 grid(a_height/32, a_width/32, batch_num);

    matrix_multi_gpu<<<grid, block>>>(batch_num1, matrix_a, matrix_b, matrix_c, a_height, a_width, int a_width, b_width, a_height, b_width);
    return;
}

__global__ void matrix_multi_without_transpose_gpu(int batch_num, float *matrix_a, float *matrix_b, float *matrix_c, int height_a,
                                    int width_a, int height_b,
                                    int width_b, int height_c,
                                    int width_c, char* mask, double scale)
{
    #define LOCAL_BLOCK_SIZE 32
    __shared__ float ds_M[LOCAL_BLOCK_SIZE][LOCAL_BLOCK_SIZE];
    __shared__ float ds_N[LOCAL_BLOCK_SIZE][LOCAL_BLOCK_SIZE];
    int g_pos_a = blockIdx.z * height_a * width_a;
    int g_pos_b = blockIdx.z * height_b * width_b;
    int g_pos_c = = blockIdx.z * height_c * width_b;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * LOCAL_BLOCK_SIZE + ty;
    int Col = bx * LOCAL_BLOCK_SIZE + tx;
    float Pvalue = 0;
    for (int m = 0; m < (width_a - 1) / LOCAL_BLOCK_SIZE + 1; ++m) {
        if (Row < height_a && m * LOCAL_BLOCK_SIZE + tx < width_a) {
            ds_M[ty][tx] = matrix_a[g_pos_a + Row * width_a + m * LOCAL_BLOCK_SIZE + tx];
        } else {
            ds_M[ty][tx] = 0.0;
        }

        if (Col < height_b && m * LOCAL_BLOCK_SIZE + tx < height_b) {
            ds_N[ty][tx] = matrix_b[g_pos_b + (m * LOCAL_BLOCK_SIZE + ty) * width_b + Col];
        } else {
            ds_N[ty][tx] = 0.0;
        }
        __syncthreads();

        for (int k = 0; k < LOCAL_BLOCK_SIZE; ++k) {
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        }
        __syncthreads();
    }

    if (Row < height_c && Col < width_c) {
        matrix_c[g_pos_c + Row * width_c + Col] = Pvalue/scale;
    }
    return;
}

void matrix_multi_without_transpose(int batch_num, float* matrix_a, int a_height, int a_num, int a_width,
                                                       float* matrix_b, int b_height, int b_width,
                                                       char* mask, double scale, float* matrix_c)
{
    dim3 block(32, 32, 1);
    dim3 grid(a_height/32, a_width/32, batch_num);

    matrix_multi_without_transpose_gpu<<<grid, block>>>(batch_num1, matrix_a, matrix_b, matrix_c, a_height, a_width, int a_width, b_width, a_height, b_width, mask, scale);
    return;
}

__global__ void add_layer_norm_gpu(int batch_num, float* matrix_a, int height, int width, float* matrix_c, char* gamme, char* beta)
{
    int row = grid.z * height * width + threadIdx.x * width;
    //#0 һ���߳���һ��
    //mean
    for (int x = 0; x < width; x++) {
        mean += matrix_a[row + x];
    }
    mean = mean / width;

    //var
    for (int x = 0; x < width; x++) {
        tmp = matrix_a[row + x] - mean;
        tmp = tmp * tmp;
        var += tmp;
    }
    var = var / width;
    var += 1e-3;//avoid to too small
    var = sqrt(var);

    for (int x = 0; x < width; x++) {
        tmp = (matrix_a[row + x] - mean) / var;
        matrix_c[row + x] = (float)(gamme[x] * tmp + beta[x]);
    }

}

void add_layer_norm(int batch_num, float* matrix_a, int height, int width, float* matrix_c, char* gamme, char* beta)
{
    dim3 block(width, 1, 1);
    dim3 grid(1, 1, batch_num);
    add_layer_norm_gpu<<<grid, block>>>(int batch_num, float* matrix_a, int height, int width, float* matrix_c, char* gamme, char* beta);
    return;
}

__global__ void softmax_gpu(int batch_num, float* matrix_a, int height, int width, float* matrix_c)
{
    int row = grid.z * height * width + threadIdx.x * width;
    double max_value = 0.0, sum = 0.0;

    for (int x = 0; x < width; x++) {
        if (max_value < matrix_a[row + x])
            max_value = matrix_a[row + x];
    }
    max_value -= 1.0;

    for (int x = 0; x < width; x++) {
        sum += exp(matrix_a[row + x] - max_value);
    }

    for (int x = 0; x < width; x++) {
        matrix_c[row + x] = (float)(exp(matrix_a[row + x] - max_value) / sum);
    }
    return;
}

void softmax(int batch_num, float* matrix_a, int height, int width, float* matrix_c)
{
    dim3 block(width, 1, 1);
    dim3 grid(1, 1, batch_num);
    softmax_gpu<<<grid, block>>>(batch_num, matrix_a, height, width, matrix_c, gamme, beta);

    return;
}